/**
 * A very simple matrix multiplication example to demonstrate the difference between row-wise and column-wise matrix multiplication.
 * It doesn't handle error checking and is not optimized for performance.
*/
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void row_wise_mat_mul_kernel(int *A, int* B, int* C, unsigned int m, unsigned int n, unsigned int p) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < m) {
        for (unsigned int j = 0; j < p; j++) {
            C[row * p + j] = 0;
            for (unsigned int k = 0; k < n; k++) {
                C[row * p + j] += A[row * n + k] * B[k * p + j];
            }
        }
    }
}

__global__ void col_wise_mat_mul_kernel(int *A, int *B, int *C, unsigned int m, unsigned int n, unsigned int p) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < p) {
        for (unsigned int i = 0; i < m; i++) {
            C[i * p + col] = 0;
            for (unsigned int k = 0; k < n; k++) {
                C[i * p + col] += A[i * n + k] * B[k * p + col];
            }
        }
    }
}

void row_wise_mat_mul_gpu(int *A, int* B, int* C, unsigned int m, unsigned int n, unsigned int p) {
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * n * sizeof(int));
    hipMalloc(&d_B, n * p * sizeof(int));
    hipMalloc(&d_C, m * p * sizeof(int));

    hipMemcpy(d_A, A, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * p * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(32, 32);
    dim3 grid((p + block.x - 1) / block.x, (m + block.y - 1) / block.y);

    row_wise_mat_mul_kernel<<<grid, block>>>(d_A, d_B, d_C, m, n, p);

    hipMemcpy(C, d_C, m * p * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void col_wise_mat_mul_gpu(int *A, int* B, int* C, unsigned int m, unsigned int n, unsigned int p) {
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * n * sizeof(int));
    hipMalloc(&d_B, n * p * sizeof(int));
    hipMalloc(&d_C, m * p * sizeof(int));

    hipMemcpy(d_A, A, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * p * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(32, 32);
    dim3 grid((p + block.x - 1) / block.x, (m + block.y - 1) / block.y);

    col_wise_mat_mul_kernel<<<grid, block>>>(d_A, d_B, d_C, m, n, p);

    hipMemcpy(C, d_C, m * p * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void matmul_cpu(int *A, int *B, int *C, unsigned int m, unsigned int n, unsigned int p) {
    for (unsigned int i = 0; i < m; i++) {
        for (unsigned int j = 0; j < p; j++) {
            C[i * p + j] = 0;
            for (unsigned int k = 0; k < n; k++) {
                C[i * p + j] += A[i * n + k] * B[k * p + j];
            }
        }
    }
}

int main() {
    unsigned int M = 800;
    unsigned int N = 1000;
    unsigned int P = 800;

    int *A = new int[M * N];
    int *B = new int[N * P];
    int *C_cpu = new int[M * P];
    int *C_gpu_row = new int[M * P];
    int *C_gpu_col = new int[M * P];

    for (unsigned int i = 0; i < M * N; i++) {
        A[i] = rand() % 100;
    }

    for (unsigned int i = 0; i < N * P; i++) {
        B[i] = rand() % 100;
    }

    // Measure time for CPU
    auto start_cpu = std::chrono::high_resolution_clock::now();
    matmul_cpu(A, B, C_cpu, M, N, P);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_cpu = end_cpu - start_cpu;
    std::cout << "CPU time: " << elapsed_cpu.count() << " s" << std::endl;

    auto start_gpu_row = std::chrono::high_resolution_clock::now();
    row_wise_mat_mul_gpu(A, B, C_gpu_row, M, N, P);
    auto end_gpu_row = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_gpu_row = end_gpu_row - start_gpu_row;
    std::cout << "GPU row-wise time: " << elapsed_gpu_row.count() << " s" << std::endl;

    for (unsigned int i = 0; i < M * P; i++) {
        if (C_cpu[i] != C_gpu_row[i]) {
            std::cout << "Mismatch at index " << i << " expected " << C_cpu[i] << " got " << C_gpu_row[i] << std::endl;
            break;
        }
    }

    auto start_gpu_col = std::chrono::high_resolution_clock::now();
    col_wise_mat_mul_gpu(A, B, C_gpu_col, M, N, P);
    auto end_gpu_col = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_gpu_col = end_gpu_col - start_gpu_col;
    std::cout << "GPU col-wise time: " << elapsed_gpu_col.count() << " s" << std::endl;

    for (unsigned int i = 0; i < M * P; i++) {
        if (C_cpu[i] != C_gpu_col[i]) {
            std::cout << "Mismatch at index " << i << " expected " << C_cpu[i] << " got " << C_gpu_col[i] << std::endl;
            break;
        }
    }

    delete[] A;
    delete[] B;
    delete[] C_cpu;
    delete[] C_gpu_row;
    delete[] C_gpu_col;

    return 0;
}